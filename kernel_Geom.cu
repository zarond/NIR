#include "hip/hip_runtime.h"
//nvcc -ptx "E:\семестр 7\НИР\kernel_Geom.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64"
//nvcc -ptx "E:\семестр 7\НИР\kernel_Geom.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64" -gencode arch=compute_52,code=sm_52 -rdc=true
//nvcc -ptx "D:\семестр 7\НИР\kernel_Geom.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.28.29333\bin\Hostx64\x64" -gencode arch=compute_52,code=sm_52 -rdc=true
#include "inc\hip/hip_vector_types.h"
        
#define eps 0.000001f
#define pi 3.141592654f
        
inline __device__ float cross(float2 a, float2 b){return a.x*b.y-a.y*b.x;}

__device__ bool ComputeIntersection(float4 * Edges, const unsigned int N, float2 pos, float2 direction, float2 &newpos, float2 &newdirection, float &r,const float s_r, const float x_s, const float y_s){
    float4 chosenEdge;
    float minR=1.0e+37f;
    bool hitSource = false;
    
    float2 SourceDirection = pos - make_float2(x_s,y_s);
    float b = dot( SourceDirection, direction );
    float c = dot( SourceDirection, SourceDirection ) - s_r*s_r;
    float h = b*b - c;
    if( h>=0.0f && b<=0.0f){ // intersection with source
        h = sqrt(h);
        minR = -b-h;
        hitSource = true;
    }

    for (int i=0;i<N;++i){
        float4 edge = Edges[i];
        SourceDirection = make_float2(pos.x - edge.x,pos.y - edge.y);
        float2 edgedir = make_float2(edge.z-edge.x,edge.w-edge.y);
        float2 v3 = make_float2(-direction.y, direction.x);
        //v3 = normalize(v3);

        float dotv = dot(edgedir,v3);
        //if (abs(dotv) < eps)
        //   continue;

        float t1 = cross(edgedir,SourceDirection) / dotv;
        float t2 = dot(SourceDirection,v3) / dotv;

        if (t1 >= 0.0f && (t2 >= 0.0f && t2 <= 1.0f) && (t1<=minR))
        {
            chosenEdge = edge;
            minR=t1;
            hitSource = false;
        }
    }
    if (minR<1.0e+35f){
        r += minR;//+eps;
        if (hitSource == true) return true;
        float2 edgedir = make_float2(chosenEdge.z-chosenEdge.x,chosenEdge.w-chosenEdge.y);
        float2 n = normalize(make_float2(-edgedir.y,edgedir.x));
        newdirection = direction-2.0f*dot(n,direction)*n;
        newdirection = normalize(newdirection);
        newpos = pos+(1.0f-eps)*minR*direction;// + newdirection*eps;
        return false;
    }
    r=-1.0f;
    return false;
}        
 
__global__ void kernelRayTracing(const float *Edges_f, const unsigned int N, float *IR, const unsigned int SampleCount,const int maxReflections,const int T,const float v,const float d_t,const float x_ir,const float y_ir, const float x_s,const float y_s, const float s_r=0.1f)
{
    float4 *Edges = (float4*)Edges_f;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= SampleCount) return;
	
    float2 pos = make_float2(x_ir,y_ir);
    float phi = 2.0f*float(i)*pi/SampleCount;
    float2 direction = make_float2(cos(phi),sin(phi));

    float r=0.0f;
    float2 newpos;
    float2 newdirection;
    bool sign = true;

    i=0;
    for (;i<maxReflections;++i){
        bool hitsource = ComputeIntersection(Edges, N, pos, direction, newpos, newdirection, r ,s_r, x_s, y_s);
        if (hitsource) break;
        pos = newpos;
        direction = newdirection;
        sign=!sign;
    }
    if (i>=maxReflections) return;
    int ind = int(r/(v*d_t));
    if (ind<T && ind>=0) 
        r = (sign)? r : -r;
        float g = 2.0f*pi/SampleCount;
        g = (sign)? g : -g;
        atomicAdd(&IR[ind],g);

}