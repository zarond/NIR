#include "hip/hip_runtime.h"
//nvcc -ptx "E:\семестр 7\НИР\kernel_IS.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64"
//nvcc -ptx "E:\семестр 7\НИР\kernel_IS.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64" -gencode arch=compute_35,code=sm_35 -rdc=true
#include "inc\hip/hip_vector_types.h"
#include "inc\hip/hip_vector_types.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
        
#define eps 0.000001
#define pi 3.141592654
#define MAXCUTS 16
        
inline __device__ float2 xy(float4 a) {return make_float2(a.x,a.y);}
inline __device__ float2 zw(float4 a) {return make_float2(a.z,a.w);}
inline __device__ bool operator==(float4 a, float b) {return (a.x==b && a.y==b && a.z==b && a.w==b);}
inline __device__ bool operator<=(float2 a, float b) {return (a.x<=b && a.y<=b);}
inline __device__ float4 make_float4(float2 a, float2 b) {return make_float4(a.x,a.y,b.x,b.y);}
        
inline __device__ float cross(float2 a, float2 b){return a.x*b.y-a.y*b.x;}
inline __device__ float2 normal(float2 a){return make_float2(-a.y,a.x);}
inline __device__ float2 reflect(float2 a, float4 b){
    float2 n = normalize(zw(b)-xy(b));
    return xy(b)+(-a+xy(b))-2*dot(n,(-a+xy(b)))*n;
}

struct ISource{
    float2 pos;
    float timeoffset;
    float4 window;
    int edgeid;
    int parent_source;
    int reflections;
    int difractions;
};

__device__ ISource* Sources;
__device__ unsigned int NumberOfSources = 0;

__device__ float2 IntersectRayEdge(float2 ro, float2 direction,float4 edge){
    float2 SourceDirection = make_float2(ro.x - edge.x,ro.y - edge.y);
    float2 edgedir = make_float2(edge.z-edge.x,edge.w-edge.y);
    float2 v3 = normal(direction);//make_float2(-direction.y, direction.x);

    float dotv = dot(edgedir,v3);
    if (dot(normal(edgedir),SourceDirection) < eps || abs(dotv) < eps) // обратная сторона
        return make_float2(0.0,0.0); // возвращает неверную ориентацию

    float t1 = cross(edgedir,SourceDirection) / dotv;
    float t2 = dot(SourceDirection,v3) / dotv;

    //if (t1 >= 0.0 && (t2 >= 0.0 && t2 <= 1.0) && (t1<=minR)){}
    return make_float2(t1,t2);
}

__device__ float4 CutEdgeWithBeam(float2 ro, float4 window, float4 edge, int sourceid=-1){
    // вычисляем пересекает ли конус видимости ребро и не загорожено ли оно другими ребрами
    // сначала попадает ли ребро в конус видимости вообще
    bool CanBeSeen = (sourceid==0); // если id == 0 то значит - источник и распространяется во все стороны
    float2 t_1 = IntersectRayEdge(/*pos*/xy(window), xy(window) - ro, edge);
    float2 t_2 = IntersectRayEdge(/*pos*/zw(window), zw(window) - ro, edge);
    
    bool notseen =         
    !(t_1.x >= 0.0 && t_1.y >= 0.0 && t_1.y <= 1.0) && !(t_2.x >= 0.0 && t_2.y >= 0.0 && t_2.y <= 1.0) //ни одна, ни вторая не пересекают
    && !((t_2.y <= 0 && t_2.x >= 0 && (t_1.x >= 0 && t_1.y >= 1 || t_1.x <= 0 || t_1.x >= 0 && t_1.y <= t_2.y )) // и не один из этих случаев
    || (t_1.y >= 1 && t_1.x >= 0 && ( t_2.x <= 0 || t_2.x >= 0 && t_2.y >= t_1.y )) 
    || (t_1.x<=0 && t_2.x<=0 && t_2.y >= t_1.y))
    || (t_1.x==0 && t_2.x==0 && t_1.y==0 && t_2.y==0) || (dot(normal(zw(edge)-xy(edge)),ro-xy(edge)) < eps);

    CanBeSeen = CanBeSeen || !notseen;
    if (CanBeSeen==false) {
        return make_float4(0.0); // возврат, что не пересекается вообще //test
    }

    float4 newwindow = edge;
    float2 edgedir = make_float2(edge.z-edge.x,edge.w-edge.y);
    if (t_2.y>0 && t_2.x>0 && sourceid!=0) xy(newwindow) = xy(edge) + edgedir*t_2.y; // обрезание ребра конусом
    if (t_1.y<1 && t_1.x>0 && sourceid!=0) zw(newwindow) = xy(edge) + edgedir*t_1.y;
    
    return newwindow;
}

__device__ void CutBeamWithEdge(float2 ro, float4 window, float4 farwindow, float4 edge, float2* cuts, int &numberOfCuts){
    if (dot(normal(zw(farwindow) - xy(farwindow)),xy(edge)-xy(farwindow))<eps 
        && dot(normal(zw(farwindow) - xy(farwindow)),zw(edge)-xy(farwindow))<eps 
        || dot(edge,edge)<=eps)
        // ребро находится за farwindow (оба конца) или перед window (это отсеивается в cutEdgeWithBeam) или размер ребро - ноль
        return;
    // если подрезает с боков
    // записываем в cuts проекцию на farwindow (в процентах)
    float2 t1 = IntersectRayEdge(ro, zw(edge)-ro, farwindow);
    float2 t2 = IntersectRayEdge(ro, xy(edge)-ro, farwindow);
    //if (t1.x>0){ t1.y = max(t1.y,0.0); t1.y = min(t1.y,1.0);}
    if (t1.x>0){ t1.y = max(t1.y,0.0); t1.y = min(t1.y,1.0);}
    else if (t1.x<0) {return;t1.y = 1;}
    if (t2.x>0){ t2.y = max(t2.y,0.0); t2.y = min(t2.y,1.0);}
    //if (t2.x>0){ t2.y = max(t2.y,0.0); t2.y = min(t2.y,1.0);}
    else if (t2.x<0) {return;t2.y = 0;}

    if (t1.y - t2.y >= eps && t1.x!=0 && t2.x!=0) // не равны друг другу (можно было и t1.y != t2.y)
        cuts[numberOfCuts++] = make_float2(t2.y,t1.y); 
}

__device__ void sortCuts(float2* A,int N){
    for ( int i = 1;i < N;++i){
        float2 x = A[i];
        int j = i - 1;
        for (;j >= 0 && A[j].x > x.x;--j)
        //while (j >= 0 && A[j] > x)
        {
            A[j+1] = A[j];
            //j = j - 1;
        }
        A[j+1] = x;
    }
}

__device__ void AddNewSource(/*unsigned int &NumberOfSources, */const ISource source,float4 newwindow/*, ISource* Sources*/,unsigned int MaxSources, int edgeid, int sourceid){
    ISource newsource = {
        reflect(source.pos, newwindow),
        source.timeoffset,
        newwindow,
        edgeid,
        sourceid,
        source.reflections+1,
        source.difractions,
    };
    int idx = atomicAdd(&NumberOfSources, 1);
    if (idx<MaxSources)
        Sources[idx] = newsource;
}

__global__ void ComputeNewIS(const float4* Edges,/*ISource* Sources,*/ const unsigned int N, 
        const unsigned int MaxSources,const unsigned int CurrentNumberOfSources /*,unsigned int &CurrentSource, unsigned int &NumberOfSources*/){
    __shared__ float4 shared[1024]; // общая память под обрезание конусов ребрами
            
    int sourceid = blockIdx.x + CurrentNumberOfSources; //???
    int edgeid = threadIdx.x;
    float4 edge = Edges[edgeid];
    ISource source = Sources[sourceid];
    
    float4 newwindow = CutEdgeWithBeam(source.pos, source.window, edge, sourceid);
    shared[edgeid]=newwindow;
    __syncthreads();
    if (newwindow==0.0) {
        return; // возврат, что не пересекается вообще
    }
    //----------------------------------------------------------
    
    float2 cuts[MAXCUTS];// = new float2[MAXCUTS]; // максимум можно разрезать конус на MAXCUTS частей
    int numberOfCuts = 0;
    for (int i=0;i<N && numberOfCuts<MAXCUTS;++i){ // рассчитать заслон от других ребер
        edge = Edges[i];
        if (i==edgeid || i==source.edgeid) continue; // чтобы edge и window сами себя не отсекали ..? надо улучшить 
        CutBeamWithEdge(source.pos, source.window, newwindow, shared[i], cuts, numberOfCuts);
    }
    //numberOfCuts = 2; cuts[1]=make_float2(0.0,0.6);cuts[0]=make_float2(0.5,0.7);

    // отсортируем по cuts.x
    sortCuts(cuts,numberOfCuts);
    float2 lr=make_float2(0.0,0.0); // границы
        
    float* cutsSingle = (float*) cuts;
    int2 indexes = make_int2(0); 
    for (int i = 0/*2*i + ((cuts[i].x>0)? 0 : 1)*/;i<2*numberOfCuts;++i){        
        if (lr.y - lr.x > 0 && (indexes.x % 2 == 1 && indexes.y % 2 == 0 || indexes.x==0 && indexes.y==0)){
            // добавлять новые мнимые источники
            float4 cutwindow = make_float4(xy(newwindow)+lr.x*(zw(newwindow)-xy(newwindow)),xy(newwindow)+lr.y*(zw(newwindow)-xy(newwindow))); 
            AddNewSource(/*NumberOfSources,*/source,cutwindow/*,Sources*/,MaxSources,edgeid,sourceid);
        }
        lr.x = lr.y;
        indexes.x = indexes.y;
        if (cutsSingle[i] >= lr.y){
            lr.y =cutsSingle[i];
            indexes.y = i;
        }
    }
    if (lr.y<1){ //последний промежуток
        float4 cutwindow = make_float4(xy(newwindow)+lr.y*(zw(newwindow)-xy(newwindow)),zw(newwindow)); 
        AddNewSource(/*NumberOfSources,*/source,cutwindow/*,Sources*/,MaxSources,edgeid,sourceid);
    }

}

__global__ 
void kernelIS(const float4 * Edges, ISource* d_Sources, const unsigned int N, const unsigned int MaxSources, const float x_s,const float y_s)
{
    Sources = d_Sources;
    /*unsigned int */NumberOfSources = 1;
    unsigned int CurrentSource = 0;
    Sources[0]={make_float2(x_s,y_s),0.0,make_float4(0.0),-1,-1,0,0};
    
    for (;NumberOfSources<MaxSources;){
    	dim3 blockDim(N, 1, 1); // при условии что N<1024
        dim3 gridDim((NumberOfSources-CurrentSource), 1, 1);
        unsigned int tmp = NumberOfSources;
        ComputeNewIS<<<gridDim,blockDim>>>(Edges,/* Sources,*/ N, MaxSources, CurrentSource /*,CurrentSource, NumberOfSources*/);
        hipDeviceSynchronize();
        CurrentSource = tmp;
    }
}

void mexFunction(int n_out, mxArray *Arr_out[], int n_in, const mxArray *Arr_in[])
{
    char const * const errId = "parallel:gpu:kernelM:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";
    /* Throw an error if the input is not a GPU array. */
    if ((n_out!=1) || (n_in<5) /*|| !(mxIsGPUArray(Arr_in[0]))*/ /*|| !(mxIsGPUArray(Arr_in[3]))*/) {
       mexErrMsgIdAndTxt(errId, errMsg);
    }

    //INPUTS
    const unsigned int N = mxGetScalar(Arr_in[1]);
    const unsigned int MaxSources = mxGetScalar(Arr_in[2]);
    const float x_s = (float)*mxGetPr(Arr_in[3]);
    const float y_s = (float)*mxGetPr(Arr_in[4]);
           
    mxInitGPU();

    if (mxGetClassID(Arr_in[0]) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }
    mxGPUArray *Edges = mxGPUCopyFromMxArray(Arr_in[0]);
    const float4 *d_Edges = (float4 *) mxGPUGetData(Edges);
	ISource* d_Sources;
    ISource* host_Sources = new ISource[MaxSources];
            
    hipMalloc((void**)&d_Sources, sizeof(ISource)*MaxSources);

    kernelIS<<<1,1>>>(d_Edges, d_Sources,/*ISource* Sources,*/ N, MaxSources, x_s, y_s);
    hipDeviceSynchronize();

    hipMemcpy( host_Sources, d_Sources, sizeof(ISource)*MaxSources,hipMemcpyDeviceToHost);

    hipFree(d_Sources);

    /* Wrap the result up as a MATLAB gpuArray for return. */
    mwSize dims[2] = {1, MaxSources};
    const char* field_names[] = {"x","y"};

    mxArray* OutArray = mxCreateDoubleMatrix(9, MaxSources, mxREAL);
    double * data = (double *) mxGetData(OutArray);
    for (int i = 0; i < MaxSources; ++i) {
        data[i*9] = (double)host_Sources[i].pos.x; 
        data[i*9+1] = (double)host_Sources[i].pos.y;
        data[i*9+2] = (double)host_Sources[i].window.x; 
        data[i*9+3] = (double)host_Sources[i].window.y;
        data[i*9+4] = (double)host_Sources[i].window.z; 
        data[i*9+5] = (double)host_Sources[i].window.w;
        data[i*9+6] = (double)host_Sources[i].edgeid;
        data[i*9+7] = (double)host_Sources[i].parent_source;
        data[i*9+8] = (double)host_Sources[i].reflections;
    }


    hipHostFree( host_Sources );

    Arr_out[0] = OutArray;
}