#include "hip/hip_runtime.h"
//nvcc -ptx "E:\семестр 7\НИР\kernel_visibility.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64"
//nvcc -ptx "E:\семестр 7\НИР\kernel_visibility.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64" -gencode arch=compute_35,code=sm_35 -rdc=true
#include "inc\hip/hip_vector_types.h"
        
#define eps 0.000001
#define pi 3.141592654
        
inline __device__ float cross(float2 a, float2 b){return a.x*b.y-a.y*b.x;}
inline __device__ float2 xy(float4 a) {return make_float2(a.x,a.y);}
inline __device__ float2 zw(float4 a) {return make_float2(a.z,a.w);}
inline __device__ bool operator==(float4 a, float b) {return (a.x==b && a.y==b && a.z==b && a.w==b);}
inline __device__ bool operator<=(float2 a, float b) {return (a.x<=b && a.y<=b);}
inline __device__ float4 make_float4(float2 a, float2 b) {return make_float4(a.x,a.y,b.x,b.y);}      
inline __device__ float2 normal(float2 a){return make_float2(-a.y,a.x);}

struct ISource{
    float2 pos;
    float timeoffset;
    float4 window;
    int edgeid;
    int parent_source;
    int reflections;
    int difractions;
};

__device__ float2 IntersectRayEdge(float2 ro, float2 direction,float4 edge){
    float2 SourceDirection = make_float2(ro.x - edge.x,ro.y - edge.y);
    float2 edgedir = make_float2(edge.z-edge.x,edge.w-edge.y);
    float2 v3 = normal(direction);//make_float2(-direction.y, direction.x);

    float dotv = dot(edgedir,v3);
    if (dot(normal(edgedir),SourceDirection) < eps || abs(dotv) < eps) // обратная сторона
        return make_float2(0.0,0.0); // возвращает неверную ориентацию

    float t1 = cross(edgedir,SourceDirection) / dotv;
    float t2 = dot(SourceDirection,v3) / dotv;

    //if (t1 >= 0.0 && (t2 >= 0.0 && t2 <= 1.0) && (t1<=minR)){}
    return make_float2(t1,t2);
}

inline __device__ bool ComputeVisibility(float4 * Edges, const unsigned int N, float2 pos, float2 direction, const ISource source, float &r){
    int chosenEdgeid;
    float minR=1.0e+37;

    float2 twindow = IntersectRayEdge(pos, direction,source.window);
    if (!(twindow.x > 0.0 && (twindow.y >= 0.0 && twindow.y <= 1.0)) && source.reflections != 0){ // не пересекается с окном источника
        r=-1;
        return false;
    }
    
    for (int i=0;i<N;++i){
        float4 edge = Edges[i];
        float2 t = IntersectRayEdge(pos, direction, edge);
        if (t.x > 0.0 && (t.y >= 0.0 && t.y <= 1.0) && (t.x<=minR))
        {
            chosenEdgeid = i;
            minR=t.x;
        }
    }
    float tmp = sqrt(dot(source.pos - pos,source.pos - pos));
    if (source.reflections == 0 && minR > tmp){
        r = tmp;
        return true;
    }
    if (minR<1.0e+35 && chosenEdgeid == source.edgeid){
        r = minR;
        return true;
    }
    r=-1.0;
    return false;
}        
 
__global__ void kernelVisibility(const float *Edges_f, const unsigned int N, const float *Sources, float* visibility, float *IR, const unsigned int M,const int T,const float v,const float d_t,const float x_ir,const float y_ir)
{ // visibility возвращает расстояние по лучу от слушателя к ребру, относительно которого построен источник
    float4 *Edges = (float4*)Edges_f;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= M) return;
    ISource source = {make_float2(Sources[i*9],Sources[i*9+1]), 0,
    make_float4(Sources[i*9+2],Sources[i*9+3],Sources[i*9+4],Sources[i*9+5]),int(round(Sources[i*9+6])),int(round(Sources[i*9+7])),int(round(Sources[i*9+8])),0};

    float2 pos = make_float2(x_ir,y_ir);
    float2 direction = normalize(source.pos - pos);

    float r=-1;
    bool sign = (source.reflections % 2 == 0);

    bool hitsource = ComputeVisibility(Edges, N, pos, direction, source,r);
    visibility[i] = r;

    if (hitsource==false) return;

    r = sqrt(dot(source.pos - pos,source.pos - pos));
    int ind = int(r/(v*d_t));
    if (ind<T && ind>=0) 
        r = (sign)? r : -r;
        atomicAdd(&IR[ind],2*pi/r);

    //i = blockIdx.x * blockDim.x + threadIdx.x;
    //if (i<T) IR[i]=r;
}