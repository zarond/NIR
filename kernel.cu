#include "hip/hip_runtime.h"
//nvcc -ptx "E:\семестр 7\НИР\kernel.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64"
//nvcc -ptx "E:\семестр 7\НИР\kernel.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64" -gencode arch=compute_35,code=sm_35 -rdc=true
#include "mex.h"
#include "gpu/mxGPUArray.h"
        

__global__ void kernel(float *U, const unsigned int N,const unsigned int M,const int t,const float v,const float d_x,const float d_t, const float b=20)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N*M) return;
    unsigned int x = i % M;
    unsigned int y = i / M;
    if (x+1>=M || y+1>=N || x<1 || y<1) {U[(x+y*M)*4+(t+1)%3]=0.0;return;}
	
    float val;
    if (U[(x+y*M)*4+3] > 0.0){
        val = -4*U[(x+y*M)*4+t%3];
        val+= (U[(x-1+y*M)*4+3]>0.0)? U[(x-1+y*M)*4+t%3]:0;//U[(x+1+y*M)*4+t%3];
        val+= (U[(x+1+y*M)*4+3]>0.0)? U[(x+1+y*M)*4+t%3]:0;//U[(x-1+y*M)*4+t%3];
        val+= (U[(x+M+y*M)*4+3]>0.0)? U[(x+M+y*M)*4+t%3]:0;//U[(x-M+y*M)*4+t%3];
        val+= (U[(x-M+y*M)*4+3]>0.0)? U[(x-M+y*M)*4+t%3]:0;//U[(x+M+y*M)*4+t%3];
        val*=(U[(x+y*M)*4+3]*U[(x+y*M)*4+3])*v*v*d_t*d_t/(d_x*d_x);  // ???
        //float b=20;
        val += 2*U[(x+y*M)*4+t%3]-U[(x+y*M)*4+(t-1)%3]*(1-d_t*b*0.5);
        val/=(1+d_t*b*0.5);
    } else {val = 0.0;}
    U[(x+y*M)*4+(t+1)%3]=val;
    //}

}

__global__ 
void kernelM(float *U, float* IR, const unsigned int N,const unsigned int M,const int t,
        const int T,const float v,const float d_x,const float d_t, const int x_ir,const int y_ir)
{
    dim3 blockDim(1024, 1, 1);
    dim3 gridDim(N*M/1024, 1, 1);
    for (int time=t;time<T;++time){
        kernel<<<gridDim,blockDim>>>(U,N,M,time+1,v,d_x,d_t);
        //__syncthreads();
        hipDeviceSynchronize();
        IR[time]=U[(x_ir+y_ir*M)*4+(time+2)%3];
    }
}

__global__ void kernelAndSetIR(float *U, float *IR, const unsigned int N,const unsigned int M,const int t,const float v,const float d_x,const float d_t,const int x_ir,const int y_ir, const float b=20)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N*M) return;
    unsigned int x = i % M;
    unsigned int y = i / M;
    if (x+1>=M || y+1>=N || x<1 || y<1) {U[(x+y*M)*4+(t+1)%3]=0.0;return;}
	
    float val;
    if (U[(x+y*M)*4+3] > 0.0){
        val = -4*U[(x+y*M)*4+t%3];
        val+= (U[(x-1+y*M)*4+3]>0.0)? U[(x-1+y*M)*4+t%3]:0;//U[(x+1+y*M)*4+t%3];
        val+= (U[(x+1+y*M)*4+3]>0.0)? U[(x+1+y*M)*4+t%3]:0;//U[(x-1+y*M)*4+t%3];
        val+= (U[(x+M+y*M)*4+3]>0.0)? U[(x+M+y*M)*4+t%3]:0;//U[(x-M+y*M)*4+t%3];
        val+= (U[(x-M+y*M)*4+3]>0.0)? U[(x-M+y*M)*4+t%3]:0;//U[(x+M+y*M)*4+t%3];
        val*=(U[(x+y*M)*4+3]*U[(x+y*M)*4+3])*v*v*d_t*d_t/(d_x*d_x); // ???
        //float b=20;
        val += 2*U[(x+y*M)*4+t%3]-U[(x+y*M)*4+(t-1)%3]*(1-d_t*b*0.5);
        val/=(1+d_t*b*0.5);
    } else {val = 0.0;}
    U[(x+y*M)*4+(t+1)%3]=val;
    if (x==x_ir && y==y_ir) IR[t-1]=val; // (t-1) because we run with t+1
    //}

}

__global__ void kernelAndSetIRAndSource(float *U, float *IR, const unsigned int N,const unsigned int M,const int t,const float v,const float d_x,const float d_t,const int x_ir,const int y_ir, const int x_s,const int y_s,float* F,const float b=20)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N*M) return;
    unsigned int x = i % M;
    unsigned int y = i / M;
    if (x+1>=M || y+1>=N || x<1 || y<1) {U[(x+y*M)*4+(t+1)%3]=0.0;return;}
	
    float val;
    if (U[(x+y*M)*4+3] > 0.0){
        val = -4*U[(x+y*M)*4+t%3];
        val+= (U[(x-1+y*M)*4+3]>0.0)? U[(x-1+y*M)*4+t%3]:0;//U[(x+1+y*M)*4+t%3];
        val+= (U[(x+1+y*M)*4+3]>0.0)? U[(x+1+y*M)*4+t%3]:0;//U[(x-1+y*M)*4+t%3];
        val+= (U[(x+M+y*M)*4+3]>0.0)? U[(x+M+y*M)*4+t%3]:0;//U[(x-M+y*M)*4+t%3];
        val+= (U[(x-M+y*M)*4+3]>0.0)? U[(x-M+y*M)*4+t%3]:0;//U[(x+M+y*M)*4+t%3];
        if (x==x_s && y==y_s) val+=F[t-1]; // Source
        val*=(U[(x+y*M)*4+3]*U[(x+y*M)*4+3])*v*v*d_t*d_t/(d_x*d_x); // ???
        //float b=20;
        val += 2*U[(x+y*M)*4+t%3]-U[(x+y*M)*4+(t-1)%3]*(1-d_t*b*0.5);
        val/=(1+d_t*b*0.5);
    } else {val = 0.0;}
    U[(x+y*M)*4+(t+1)%3]=val;
    if (x==x_ir && y==y_ir) IR[t-1]=val; // (t-1) because we run with t+1
    //}

}

        
void mexFunction(int n_out, mxArray *Arr_out[], int n_in, const mxArray *Arr_in[])
{
    char const * const errId = "parallel:gpu:kernelM:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";
    /* Throw an error if the input is not a GPU array. */
    if ((n_out!=2) || (n_in<11) || !(mxIsGPUArray(Arr_in[0])) || !(mxIsGPUArray(Arr_in[1]))) {
       mexErrMsgIdAndTxt(errId, errMsg);
    }

    /* Declare all variables.*/
    mxGPUArray *Ugpu;
    mxGPUArray *IRgpu;
    mxGPUArray *Fgpu;
    float *d_U;
    float *d_IR;
    float *d_F;

    //INPUTS
    const unsigned int N=mxGetScalar(Arr_in[2]);
    const unsigned int M=mxGetScalar(Arr_in[3]);
    const int t=mxGetScalar(Arr_in[4]);
    const int T=mxGetScalar(Arr_in[5]);
    const float v=(float)*mxGetPr(Arr_in[6]);
    const float d_x=(float)*mxGetPr(Arr_in[7]);
    const float d_t=(float)*mxGetPr(Arr_in[8]);
    const int x_ir=mxGetScalar(Arr_in[9]);
    const int y_ir=mxGetScalar(Arr_in[10]);
    int mode,x_s,y_s;
    float b;
    if (n_in>=12) mode=mxGetScalar(Arr_in[11]);
    else mode=1;
    if (n_in>=13) b=(float)*mxGetPr(Arr_in[12]);
    if (n_in>=16) {
        x_s = mxGetScalar(Arr_in[13]);
        y_s = mxGetScalar(Arr_in[14]);
    }
           

    mxInitGPU();

    
    Ugpu = mxGPUCopyFromMxArray(Arr_in[0]);
    IRgpu = mxGPUCopyFromMxArray(Arr_in[1]);
    if (n_in>=16) {
        Fgpu = mxGPUCopyFromMxArray(Arr_in[15]);
        d_F = (float *)(mxGPUGetData(Fgpu));
    }

    if ((mxGPUGetClassID(Ugpu) != mxSINGLE_CLASS) || (mxGPUGetClassID(IRgpu) != mxSINGLE_CLASS)) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    d_U = (float *)(mxGPUGetData(Ugpu));
    d_IR = (float *)(mxGPUGetData(IRgpu));

    if (mode==0){
        kernelM<<<1, 1>>>(d_U, d_IR, N, M, t, T, v, d_x, d_t,x_ir,y_ir);
    } 
    else if (mode==1) {
        dim3 blockDim(1024, 1, 1);
        dim3 gridDim(N*M/1024, 1, 1);
        for (int time=t;time<T;++time){
            kernelAndSetIR<<<gridDim,blockDim>>>(d_U,d_IR,N,M,time+1,v,d_x,d_t,x_ir,y_ir,b);
            hipDeviceSynchronize();
        }
    } else if (mode==2 || mode==3){
        dim3 blockDim(1024, 1, 1);
        dim3 gridDim(N*M/1024, 1, 1);
        for (int time=t;time<T;++time){
            kernelAndSetIRAndSource<<<gridDim,blockDim>>>(d_U,d_IR,N,M,time+1,v,d_x,d_t,x_ir,y_ir,x_s,y_s,d_F,b);
            hipDeviceSynchronize();
        }
    }

    hipDeviceSynchronize();

    /* Wrap the result up as a MATLAB gpuArray for return. */
    Arr_out[0] = mxGPUCreateMxArrayOnGPU(Ugpu);
    Arr_out[1] = mxGPUCreateMxArrayOnGPU(IRgpu);

    if (n_in>=16) {
        mxGPUDestroyGPUArray(Fgpu);
    }
}
