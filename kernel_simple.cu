
#include <hip/hip_runtime.h>
//nvcc -ptx "E:\семестр 7\НИР\kernel.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64"
//nvcc -ptx "E:\семестр 7\НИР\kernel.cu" -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64" -gencode arch=compute_35,code=sm_35 -rdc=true
__device__ float getneighbor(float3 *A, const unsigned int N,const unsigned int M, const unsigned int x, const unsigned int y){
    if (A[x+y*M].z == 0.0) {
        return A[x+y*M].x;
    }
    return 0.0;//A[x+y*M].x;
}
__global__ void kernel(float *U, const unsigned int N,const unsigned int M,const int t,const float v,const float d_x,const float d_t, const float b=20)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N*M) return;
    unsigned int x = i % M;
    unsigned int y = i / M;
    if (x+1>=M || y+1>=N || x<1 || y<1) {U[(x+y*M)*4+(t+1)%3]=0.0;return;}
	
    float val;
    if (U[(x+y*M)*4+3] > 0.0){
        val = -4*U[(x+y*M)*4+t%3];
        val+= (U[(x-1+y*M)*4+3]>0.0)? U[(x-1+y*M)*4+t%3]:0;//U[(x+1+y*M)*4+t%3];
        val+= (U[(x+1+y*M)*4+3]>0.0)? U[(x+1+y*M)*4+t%3]:0;//U[(x-1+y*M)*4+t%3];
        val+= (U[(x+M+y*M)*4+3]>0.0)? U[(x+M+y*M)*4+t%3]:0;//U[(x-M+y*M)*4+t%3];
        val+= (U[(x-M+y*M)*4+3]>0.0)? U[(x-M+y*M)*4+t%3]:0;//U[(x+M+y*M)*4+t%3];
        val*=(U[(x+y*M)*4+3]*U[(x+y*M)*4+3])*v*v*d_t*d_t/(d_x*d_x); // ???
        //float b=20;
        val += 2*U[(x+y*M)*4+t%3]-U[(x+y*M)*4+(t-1)%3]*(1-d_t*b*0.5);
        val/=(1+d_t*b*0.5);
    } else {val = 0.0;}
    U[(x+y*M)*4+(t+1)%3]=val;
}

__global__ void kernelAndSetIR(float *U, float *IR, const unsigned int N,const unsigned int M,const int t,const float v,const float d_x,const float d_t,const int x_ir,const int y_ir, const float b=20)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N*M) return;
    unsigned int x = i % M;
    unsigned int y = i / M;
    if (x+1>=M || y+1>=N || x<1 || y<1) {U[(x+y*M)*4+(t+1)%3]=0.0;return;}
	
    float val;
    if (U[(x+y*M)*4+3] > 0.0){
        val = -4*U[(x+y*M)*4+t%3];
        val+= (U[(x-1+y*M)*4+3]>0.0)? U[(x-1+y*M)*4+t%3]:0;//U[(x+1+y*M)*4+t%3];
        val+= (U[(x+1+y*M)*4+3]>0.0)? U[(x+1+y*M)*4+t%3]:0;//U[(x-1+y*M)*4+t%3];
        val+= (U[(x+M+y*M)*4+3]>0.0)? U[(x+M+y*M)*4+t%3]:0;//U[(x-M+y*M)*4+t%3];
        val+= (U[(x-M+y*M)*4+3]>0.0)? U[(x-M+y*M)*4+t%3]:0;//U[(x+M+y*M)*4+t%3];
        val*=(U[(x+y*M)*4+3]*U[(x+y*M)*4+3])*v*v*d_t*d_t/(d_x*d_x); // ???
        //float b=20;
        val += 2*U[(x+y*M)*4+t%3]-U[(x+y*M)*4+(t-1)%3]*(1-d_t*b*0.5);
        val/=(1+d_t*b*0.5);
    } else {val = 0.0;}
    U[(x+y*M)*4+(t+1)%3]=val;
    if (x==x_ir && y==y_ir) IR[t-1]=val; // (t-1) because we run with t+1
}

__global__ void kernelAndSetIRAndSource(float *U, float *IR, const unsigned int N,const unsigned int M,const int t,const float v,const float d_x,const float d_t,const int x_ir,const int y_ir, const int x_s,const int y_s,float* F,const float b=20)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N*M) return;
    unsigned int x = i % M;
    unsigned int y = i / M;
    if (x+1>=M || y+1>=N || x<1 || y<1) {U[(x+y*M)*4+(t+1)%3]=0.0;return;}
	
    float val;
    if (U[(x+y*M)*4+3] > 0.0){
        val = -4*U[(x+y*M)*4+t%3];
        val+= (U[(x-1+y*M)*4+3]>0.0)? U[(x-1+y*M)*4+t%3]:0;//U[(x+1+y*M)*4+t%3];
        val+= (U[(x+1+y*M)*4+3]>0.0)? U[(x+1+y*M)*4+t%3]:0;//U[(x-1+y*M)*4+t%3];
        val+= (U[(x+M+y*M)*4+3]>0.0)? U[(x+M+y*M)*4+t%3]:0;//U[(x-M+y*M)*4+t%3];
        val+= (U[(x-M+y*M)*4+3]>0.0)? U[(x-M+y*M)*4+t%3]:0;//U[(x+M+y*M)*4+t%3];
        if (x==x_s && y==y_s) val+=F[t-1]; // Source
        val*=(U[(x+y*M)*4+3]*U[(x+y*M)*4+3])*v*v*d_t*d_t/(d_x*d_x); // ???
        //float b=20;
        val += 2*U[(x+y*M)*4+t%3]-U[(x+y*M)*4+(t-1)%3]*(1-d_t*b*0.5);
        val/=(1+d_t*b*0.5);
    } else {val = 0.0;}
    U[(x+y*M)*4+(t+1)%3]=val;
    if (x==x_ir && y==y_ir) IR[t-1]=val; // (t-1) because we run with t+1
}

__global__ void kernel2(float *U, float *U1, const unsigned int N,const unsigned int M,const float v,const float d_x,const float d_t)
{
    float3 *A=(float3*) U;
    float3 *A1=(float3*) U1;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N*M) return;
    unsigned int x = i % M;
    unsigned int y = i / M;
    if (x+1>=M || y+1>=N || x<1 || y<1) {    A1[x+y*M] = A[x+y*M]; return;}
	//A[i].x = i;
    float pos = A[x+y*M].x;
    float vel = A[x+y*M].y;
    float wall = A[x+y*M].z;
    if (wall == 0.0){
        float m = A[x+1+y*M].x;//getneighbor(A,N,M,x+1,y);
        m += A[x-1+y*M].x;//getneighbor(A,N,M,x-1,y);
        m += A[x+y*M+M].x;//getneighbor(A,N,M,x,y+1);
        m += A[x+y*M-M].x;//getneighbor(A,N,M,x,y-1);
        m *= .25;
        vel = 1.5*(1.0-wall)*(m-pos)+vel;
        pos=pos+vel;
    } else {pos = 0.0;vel = 0.0;}
    A1[x+y*M].x = pos;//+0.1;
    A1[x+y*M].y = vel;
    A1[x+y*M].z = wall;
}

// __global__ void kernel2(float *U, const unsigned int N,const unsigned int M,int t,const float v,const float d_x,const float d_t)
// {
//     float3 *A1=(float3*) U;
//     float3 *A2=(float3*) (U+1);
//     float3 *A3=(float3*) (U+2);
//     float3 *wall=(float3*) (U+3);
//     float3* A[3] = {A1,A2,A3};
// 	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	if (i >= N*M) return;
//     unsigned int x = i % M;
//     unsigned int y = i / M;
//     if (x+1>=M || y+1>=N || x<1 || y<1) {return;}
// 
//     //for (t:=0;t<1000;++t){
//     float val;
//     if (wall[x+y*M].x == 0.0){
//         val = A[t%3][x-1+y*M].x-4*A[t%3][x+y*M].x+A[t%3][x+1+y*M].x+A[t%3][x+y*M+M].x+A[t%3][x+y*M-M].x;
//         val*=v*v*d_t*d_t/(d_x*d_x);
//         val += 2*A[t%3][x+y*M].x-A[(t-1)%3][x+y*M].x;
//     } else {val = 0.0;}
//     A[(t+1)%3][x+y*M].x=val;
// 
// }

